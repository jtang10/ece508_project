#include "hip/hip_runtime.h"
#include "coo-impl.hpp"
#include <iostream>
#include "modifiedfilereader.cpp"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#define NUM_BLOCKS 4
__device__ uint64_t triangles_buffer_offset=0;

struct is_positive
{
    __host__ __device__
        bool operator()(int x)
    {
        return (x > 0);
    }
};

struct is_not_m1
{
    __host__ __device__
        bool operator()(int x)
    {
        return (x != -1);
    }
};

/*
 * Count triangle on each edge, store only on the lowest edge.
 */
__global__ void triangle_count(int* edgeDsts, int* rowPtrs, int* edgeSrcs, uint64_t num_rows, uint64_t num_edges, int* triangles_count) {
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	if (tid < num_edges) {
		for (int i=tid;i<num_edges;i+=blockDim.x*NUM_BLOCKS) {
			int i_src=0;
			int i_dst=0;
			int srcSize=rowPtrs[edgeSrcs[i]+1]-rowPtrs[edgeSrcs[i]];
			int destSize=rowPtrs[edgeDsts[i]+1]-rowPtrs[edgeDsts[i]];
			int tricount=0;
			while (i_src<srcSize&&i_dst<destSize) {
				if (edgeDsts[rowPtrs[edgeSrcs[i]]+i_src]<edgeDsts[rowPtrs[edgeDsts[i]]+i_dst]) {
					++i_src;
				}
				else if (edgeDsts[rowPtrs[edgeSrcs[i]]+i_src]>edgeDsts[rowPtrs[edgeDsts[i]]+i_dst]) {
					++i_dst;
				}
				else {
					++i_src;
					++i_dst;
					++tricount;
				}
			}
			triangles_count[i]=tricount;
		}
	}
}
__global__ void triangle_write(int* edgeDsts, int* rowPtrs, int* edgeSrcs, uint64_t num_rows, uint64_t num_edges, int* triangles_buffer, int* triangles_offsets) {
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	for (int i=tid;i<num_edges;i+=blockDim.x*NUM_BLOCKS) {
		int i_src=0;
		int i_dst=0;
		int srcSize=rowPtrs[edgeSrcs[i]+1]-rowPtrs[edgeSrcs[i]];
		int destSize=rowPtrs[edgeDsts[i]+1]-rowPtrs[edgeDsts[i]];
		int triangle_offset=triangles_offsets[i];
		while (i_src<srcSize&&i_dst<destSize) {
			if (edgeDsts[rowPtrs[edgeSrcs[i]]+i_src]<edgeDsts[rowPtrs[edgeDsts[i]]+i_dst]) {
				++i_src;
			}
			else if (edgeDsts[rowPtrs[edgeSrcs[i]]+i_src]>edgeDsts[rowPtrs[edgeDsts[i]]+i_dst]) {
				++i_dst;
			}
			else {
				++i_src;
				++i_dst;
				triangles_buffer[triangle_offset]=rowPtrs[edgeSrcs[i]]+i_src;
				triangles_buffer[triangle_offset+1]=rowPtrs[edgeDsts[i]]+i_dst;
				triangle_offset+=2;
			}
		}
	}
}
__global__ void triangle_scan(uint64_t num_edges, int* triangles_count, int* triangles_offsets) {
	int tid=threadIdx.x;
	int offset=1;
	for (int i=tid;i<num_edges;i+=blockDim.x) {
		triangles_offsets[i+1]=2*triangles_count[i];
	}
	__syncthreads();
	while (offset<num_edges) {
		for (int i=tid;i<num_edges;i+=blockDim.x) {
			if (i+offset<num_edges) {
				triangles_offsets[i+offset+1]+=triangles_offsets[i+1];
			}
		}
		offset*=2;
		__syncthreads();
	}
}
__global__ void truss_decomposition(int* edgeDsts, int* rowPtrs, int* edgeSrcs, uint64_t num_rows, uint64_t num_edges,int* triangles_counts, int* triangles_offsets, int* triangles_buffer,int* edge_exists, int* new_deletes, int k) {
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int local_edge_exists=0, local_new_deletes=0;
	for (int i=tid;i<num_edges;i+=blockDim.x*NUM_BLOCKS) {
		int tricount=atomicAdd(triangles_counts+i,0);
		if (tricount<(k-2)&&tricount>0) {
			triangles_counts[i]=0;
			local_new_deletes=1;
			for (int j=triangles_offsets[i];j!=triangles_offsets[i+1];j+=2) {
				int e1=triangles_buffer[j];
				int e2=triangles_buffer[j+1];
				for (int iter=triangles_offsets[e1];iter!=triangles_offsets[e1+1];iter+=2) {
					int ea=atomicAdd(triangles_buffer+iter,0);
					int eb=atomicAdd(triangles_buffer+iter+1,0);
					if (ea==i||eb==i) {
						atomicCAS(triangles_buffer+iter,ea,-1);
						atomicCAS(triangles_buffer+iter,eb,-1);
					}
				}
				for (int iter=triangles_offsets[e2];iter!=triangles_offsets[e2+1];iter+=2) {
					int ea=atomicAdd(triangles_buffer+iter,0);
					int eb=atomicAdd(triangles_buffer+iter+1,0);
					if (ea==i||eb==i) {
						atomicCAS(triangles_buffer+iter,ea,-1);
						atomicCAS(triangles_buffer+iter,eb,-1);
					}
				}
				atomicAdd(&triangles_counts[e1],-1);
				atomicAdd(&triangles_counts[e2],-1);
			}
		}
		else if(tricount>0) {
			local_edge_exists=1;
		}
	}
	atomicCAS(edge_exists,0,local_edge_exists);
	atomicCAS(new_deletes,0,local_new_deletes);
}

void truss_wrapper(COOView<int> graph) {
	int numThreadsPerBlock=128;
	const int* edgeDsts=graph.col_ind();
	const int* rowPtrs=graph.row_ptr();
	const int* edgeSrcs=graph.row_ind();
	uint64_t num_rows=graph.num_rows();
	uint64_t num_edges=graph.nnz();

	std::cout << "Edge List" << std::endl;
	for (int i = 0; i < num_edges; ++i) {
		std::cout << edgeSrcs[i] << ' ' << edgeDsts[i] << std::endl;;
	}
	
	std::cout << "Row Pointer" << std::endl;
	for (int i = 0; i < num_rows+1; ++i) {
		std::cout << rowPtrs[i] << ' ';
	}
	std::cout << std::endl;

	int* edgeDsts_d=nullptr;
	int* rowPtrs_d=nullptr;
	int* edgeSrcs_d=nullptr;
	int* triangles_count=nullptr;

	//allocate necessary memory
	hipMalloc(&edgeDsts_d,num_edges*sizeof(int));
	hipMalloc(&rowPtrs_d,(num_rows+1)*sizeof(int));
	hipMalloc(&edgeSrcs_d,num_edges*sizeof(int));
	hipMallocManaged(&triangles_count,num_edges*sizeof(int));
	//copy over data
	hipMemcpy(edgeDsts_d, edgeDsts, num_edges*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(rowPtrs_d, rowPtrs, (num_rows+1)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(edgeSrcs_d, edgeSrcs, num_edges*sizeof(int),hipMemcpyHostToDevice);
	//call triangle_count
	triangle_count<<<NUM_BLOCKS, numThreadsPerBlock>>>(edgeDsts_d,rowPtrs_d,edgeSrcs_d,num_rows,num_edges,triangles_count);
	hipDeviceSynchronize();
	std::cout << "Triangle Count" << std::endl;
	for (int i = 0; i < num_edges; ++i) {
		std::cout << triangles_count[i] << ' ';
	}
	std::cout << std::endl;
	//allocate necessary memory
	int* triangles_buffer=nullptr;
	int* triangles_offsets=nullptr;
	hipMallocManaged(&triangles_offsets,(num_edges+1)*sizeof(int));
	triangle_scan<<<1, numThreadsPerBlock>>>(num_edges,triangles_count,triangles_offsets);
	/*int start=0;
	for (int i=0;i!=num_edges;++i) {
		triangles_offsets[i]=start;
		start+=2*triangles_count[i];
	}
	triangles_offsets[num_edges]=start;*/
	hipMallocManaged(&triangles_buffer,(triangles_offsets[num_edges])*sizeof(int));
	//call triangle_write
	triangle_write<<<NUM_BLOCKS, numThreadsPerBlock>>>(edgeDsts_d,rowPtrs_d, edgeSrcs_d, num_rows, num_edges, triangles_buffer, triangles_offsets);
	hipDeviceSynchronize();

	std::cout << "Triangle Write" << std::endl;
	std::cout << "triangles_buffer" << std::endl;
	for (int i = 0; i < triangles_offsets[num_edges]; ++i) {
		std::cout << triangles_buffer[i] << ' ';
	}
	std::cout << std::endl;

	std::cout << "triangles_offsets" << std::endl;
	for (int i = 0; i < triangles_offsets[num_edges]; ++i) {
		std::cout << triangles_offsets[i] << ' ';
	}
	std::cout << std::endl;

	int* edge_exists_ptr=nullptr;
	int* new_deletes_ptr=nullptr;
	int k=2;
	hipMallocManaged(&edge_exists_ptr,sizeof(int));
	hipMallocManaged(&new_deletes_ptr,sizeof(int));
	*edge_exists_ptr=1;
	*new_deletes_ptr=0;
	while (*edge_exists_ptr) {
		if (*new_deletes_ptr==0) {
			//output current graph as k-truss subgraph
			//remove edges from graph
			//adjust num_edges here
			//perform stream compaction here
			int old_num_edges=0;
			thrust::device_vector<int> tricounts(num_edges);
			thrust::device_ptr<int> triangles_count_d(triangles_count);
			thrust::device_ptr<int> triangles_offsets_d(triangles_offsets);
			auto result_end=thrust::copy_if(triangles_count_d,triangles_count_d+old_num_edges,tricounts.begin(),is_positive());
			thrust::copy(tricounts.begin(),result_end,triangles_count_d);
			triangle_scan<<<1, numThreadsPerBlock>>>(num_edges,triangles_count_d,triangles_offsets_d);
			hipDeviceSynchronize();
			int newtricountsum=thrust::reduce(triangles_count_d,triangles_count_d+num_edges);
			//
			thrust::device_vector<int> tribuffer((newtricountsum+1)*2);
			thrust::device_ptr<int> triangles_buffer_d(triangles_buffer);
			auto result_bend=thrust::copy_if(triangles_buffer_d,triangles_buffer_d+((old_num_edges+1)*2),tribuffer.begin(),is_not_m1());
			thrust::copy(tribuffer.begin(),tribuffer.end(),triangles_buffer_d);
			++k;
		}
		*edge_exists_ptr=0;
		*new_deletes_ptr=0;
		truss_decomposition<<<NUM_BLOCKS, numThreadsPerBlock>>>(edgeDsts_d, rowPtrs_d, edgeSrcs_d, num_rows, num_edges, triangles_count, triangles_offsets, triangles_buffer, edge_exists_ptr, new_deletes_ptr, k);
		hipDeviceSynchronize();
	}
	hipFree(edgeDsts_d);
	hipFree(rowPtrs_d);
	hipFree(edgeSrcs_d);
	hipFree(triangles_count);
	hipFree(triangles_offsets);
	hipFree(triangles_buffer);
	hipFree(edge_exists_ptr);
	hipFree(new_deletes_ptr);
}
int main() {
	std::vector<std::pair<int,int>> edgetemp;
	EdgeListFile elf("./data/test2.bel");
	elf.get_edges(edgetemp,8);
	COO<int> graph=COO<int>::from_edges(edgetemp.begin(),edgetemp.end());
	truss_wrapper(graph.view());
}
