#include "hip/hip_runtime.h"
#include <vector>
#include <list>
#include <cassert>
#include <algorithm>
#include "util.cpp"
#include "edge_list_file.hpp"
#include "coo-impl.hpp"
#include <chrono>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <utility>
#include <iostream> 

__global__ static void count_triangle(int32_t *triangleCount, //!< per-edge triangle counts
                                      const int32_t *const edgeSrc,         //!< node ids for edge srcs
                                      const int32_t *const edgeDst,         //!< node ids for edge dsts
                                      const int32_t *const rowPtr,          //!< source node offsets in edgeDst
                                      const int32_t numEdges                  //!< how many edges to count triangles for
) {
  int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  for(int32_t i = idx; i < numEdges; i += blockDim.x * gridDim.x) {
    // Determine the source and destination node for the edge
    int32_t u = edgeSrc[idx];
    int32_t v = edgeDst[idx];

    // Use the row pointer array to determine the start and end of the neighbor list in the column index array
    int32_t u_ptr = rowPtr[u];
    int32_t v_ptr = rowPtr[v];

    int32_t u_end = rowPtr[u + 1];
    int32_t v_end = rowPtr[v + 1];

    int32_t w1 = edgeDst[u_ptr];
    int32_t w2 = edgeDst[v_ptr];

    // Determine how many elements of those two arrays are common
    while ((u_ptr < u_end) && (v_ptr < v_end)) {
      if (w1 < w2) {
        w1 = edgeDst[++u_ptr];
      } else if (w1 > w2) {
        w2 = edgeDst[++v_ptr];
      } else {
        w1 = edgeDst[++u_ptr];
        w2 = edgeDst[++v_ptr];
        triangleCount[idx]++;
        triangleCount[u_ptr]++;
        triangleCount[v_ptr]++;
      }
    }
  }
}

__global__ static void write_triangle(int32_t *triangleOffsets, //!< per-edge triangle offsets
                                      int32_t *triangleOffCounts,
                                      int32_t *triangleBuffers1, //!< per-edge triangle buffers
                                      int32_t *triangleBuffers2, 
                                      const int32_t *const edgeSrc,         //!< node ids for edge srcs
                                      const int32_t *const edgeDst,         //!< node ids for edge dsts
                                      const int32_t *const rowPtr,          //!< source node offsets in edgeDst
                                      const int32_t numEdges                  //!< how many edges to count triangles for
) {
  int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  for(int32_t i = idx; i < numEdges; i += blockDim.x * gridDim.x) {
    // Determine the source and destination node for the edge
    int32_t u = edgeSrc[idx];
    int32_t v = edgeDst[idx];

    // Use the row pointer array to determine the start and end of the neighbor list in the column index array
    int32_t u_ptr = rowPtr[u];
    int32_t v_ptr = rowPtr[v];

    int32_t u_end = rowPtr[u + 1];
    int32_t v_end = rowPtr[v + 1];

    int32_t w1 = edgeDst[u_ptr];
    int32_t w2 = edgeDst[v_ptr];

    // Determine how many elements of those two arrays are common
    while ((u_ptr < u_end) && (v_ptr < v_end)) {
      if (w1 < w2) {
        w1 = edgeDst[++u_ptr];
      } else if (w1 > w2) {
        w2 = edgeDst[++v_ptr];
      } else {
        w1 = edgeDst[++u_ptr];
        w2 = edgeDst[++v_ptr];
        int32_t local_offset=atomicAdd(triangleOffCounts+i,1);
        triangleBuffers1[local_offset]=u_ptr;
        triangleBuffers2[local_offset]=v_ptr;
        int32_t u_offset=atomicAdd(triangleOffCounts+u,1);
        triangleBuffers1[u_offset]=v_ptr;
        triangleBuffers2[u_offset]=i;
        int32_t v_offset=atomicAdd(triangleOffCounts+v,1);
        triangleBuffers1[v_offset]=u_ptr;
        triangleBuffers2[v_offset]=i;
      }
    }
  }
}

__global__ static void truss_decompose(int32_t* triangleCounts, //!< per-edge triangle counts
                                       int32_t* triangleOffsets, //!< per-edge triangle offsets
                                       int32_t* triangleRemove, //!< per-edge triangle removes
                                       int32_t* triangleBuffers1, //!< per-edge triangle buffer
                                       int32_t* triangleBuffers2, //!< per-edge triangle buffer
                                       int* edge_exists,
                                       int* new_deletes,
                                       int32_t numEdges,
                                       int k
) {
  int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int local_edge_exists=0;
  int local_new_deletes=0;
  for (int32_t i = idx; i < numEdges; i += blockDim.x * gridDim.x) {
    int32_t my_triangle_count=triangleCounts[i];
    if (my_triangle_count<(k-2)*3&&my_triangle_count>0) { //remove edge
      local_new_deletes=1;
      atomicAdd(triangleRemove+i,my_triangle_count);
      for (int32_t iter = triangleOffsets[i];iter!=triangleOffsets[i+1];++iter) {
        atomicAdd(triangleRemove+triangleBuffers1[iter],1);
        atomicAdd(triangleRemove+triangleBuffers2[iter],1);
      }
    }
    else if (my_triangle_count>0) {
      local_edge_exists=1;
    }
  }
  atomicCAS(edge_exists,0,local_edge_exists);
  atomicCAS(new_deletes,0,local_new_deletes);
}

__global__ static void update_triangles(int32_t* triangleCounts, //!< per-edge triangle counts
                                        int32_t* triangleOffsets, //!< per-edge triangle offsets
                                        int32_t* triangleRemove, //!< per-edge triangle removes
                                        int32_t* triangleBuffers1,
                                        int32_t* triangleBuffers2, //!< per-edge triangle buffer
                                        int32_t  numEdges
) {
  int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int32_t i = idx; i < numEdges; i += blockDim.x * gridDim.x) {
    if (triangleCounts[i]<=triangleRemove[i]) { //must remove triangles associated with this edge
      for (int32_t iter = triangleOffsets[i];iter!=triangleOffsets[i+1];++iter) {
        int32_t e1=atomicAdd(&triangleBuffers1[iter],0u);
        int32_t e2=atomicAdd(&triangleBuffers2[iter],0u);
        if (e1!=ULLONG_MAX) {
          for (int32_t j=triangleOffsets[e1];j!=triangleOffsets[e1+1];++j) {
            int32_t ea=atomicAdd(&triangleBuffers1[j],0u);
            int32_t eb=atomicAdd(&triangleBuffers2[j],0u);
            if (ea==i||eb==i) {
              atomicExch(&triangleBuffers1[j],ULLONG_MAX);
              atomicExch(&triangleBuffers2[j],ULLONG_MAX);
            }
          }
        }
        if (e2!=ULLONG_MAX) {
          for (int32_t j=triangleOffsets[e2];j!=triangleOffsets[e2+1];++j) {
            int32_t ea=atomicAdd(&triangleBuffers1[j],0u);
            int32_t eb=atomicAdd(&triangleBuffers2[j],0u);
            if (ea==i||eb==i) {
              atomicExch(&triangleBuffers1[j],ULLONG_MAX);
              atomicExch(&triangleBuffers2[j],ULLONG_MAX);
            }
          }
        }
      }
      triangleCounts[i]=0;
    }
    else {
      triangleCounts[i]-=triangleRemove[i];
    }
  }
}

int main(int argc, char * argv[]) {
  std::string test_filename;	
  if (argv[1] == NULL) {
      test_filename = "./data/test3.bel";
  } else {
    test_filename = argv[1];
  }
  EdgeListFile test_file(test_filename);

  // get the total number of edges in the file.
  std::vector<EdgeTy<int32_t>> edges;
  int32_t size = getNumEdges(test_filename);
  std::cout << "Numbers of edges in the file : " << size << std::endl;

  // read the bel file into the EdgeListFile
  int32_t numEdge = test_file.get_edges(edges, size);
  std::cout << "Confirmed read edges: " << numEdge << std::endl;
  
  COO<int32_t> coo_test = COO<int32_t>::from_edges<std::vector<EdgeTy<int32_t>>::iterator>(edges.begin(), edges.end());
  COOView<int32_t> test_view = coo_test.view();
  
  int32_t numEdges = test_view.nnz();
  int32_t numRows = test_view.num_rows();
  // vector<int32_t> triangleCount(numEdges);  // keep track of the number of triangles for each edge
  // vector<vector<pair<int32_t, int32_t>>> triangleList(numEdges); // keep track of the triangle edges for each edge
  std::cout << "numEdges from nnz: " << numEdges << std::endl;

	int32_t* edgeSrc_device = nullptr;
  int32_t* edgeDst_device = nullptr;
	int32_t* rowPtr_device = nullptr;
	int32_t* triangleCount = nullptr;
  int32_t* triangleOffsets = nullptr;
  int32_t* triangleOffCounts = nullptr;
  int32_t* triangleBuffers1 = nullptr;
  int32_t* triangleBuffers2 = nullptr;
  int32_t* triangleRemove = nullptr;
  int* edge_exists = nullptr;
  int* new_deletes = nullptr;
  int k=2;

	//allocate necessary memory
	hipMalloc(&edgeSrc_device, numEdges*sizeof(int32_t));
	hipMalloc(&edgeDst_device, numEdges*sizeof(int32_t));
	hipMalloc(&rowPtr_device, (numRows+1)*sizeof(int32_t));
	hipMallocManaged(&triangleCount, numEdges*sizeof(int32_t));
  hipMallocManaged(&triangleOffsets, (numEdges+1)*sizeof(int32_t));
  hipMallocManaged(&triangleOffCounts, (numEdges+1)*sizeof(int32_t));
  hipMallocManaged(&triangleRemove, numEdges*sizeof(int32_t));
  hipMallocManaged(&edge_exists, sizeof(int));
  hipMallocManaged(&new_deletes, sizeof(int));
  *edge_exists=1;
  *new_deletes=0;
	//copy over data
	hipMemcpy(edgeSrc_device, test_view.row_ind(), numEdges*sizeof(int32_t),hipMemcpyHostToDevice);
	hipMemcpy(edgeDst_device, test_view.col_ind(), numEdges*sizeof(int32_t),hipMemcpyHostToDevice);
	hipMemcpy(rowPtr_device, test_view.row_ptr(), (numRows+1)*sizeof(int32_t),hipMemcpyHostToDevice);
  //call triangle_count
  dim3 dimBlock(512);
  dim3 dimGrid (ceil(numEdges * 1.0 / dimBlock.x));
	count_triangle<<<dimBlock, dimGrid>>>(triangleCount, edgeSrc_device, edgeDst_device, rowPtr_device, numEdges);
  hipDeviceSynchronize();
  


  thrust::device_ptr<int32_t> triangleCount_ptr(triangleCount);
  triangleOffsets[0]=0;
  thrust::device_ptr<int32_t> triangleOffsets_ptr(triangleOffsets);
  thrust::inclusive_scan(triangleCount_ptr,triangleCount_ptr+numEdges,triangleOffsets_ptr+1);
  hipDeviceSynchronize();
  thrust::device_ptr<int32_t> triangleOffCounts_ptr(triangleOffCounts);
  thrust::copy(triangleOffsets_ptr,triangleOffCounts_ptr);
  
  hipMallocManaged(&triangleBuffers1,triangleOffsets[numEdges]*sizeof(int32_t));
  hipMallocManaged(&triangleBuffers2,triangleOffsets[numEdges]*sizeof(int32_t));
  write_triangle<<<dimBlock, dimGrid>>>(triangleOffsets, triangleOffCounts, triangleBuffers1, triangleBuffers2, edgeSrc_device, edgeDst_device, rowPtr_device, numEdges);
  hipDeviceSynchronize();
  
  while (*edge_exists) {
    if (*new_deletes==0) {
      ++k;
    }
    *edge_exists=0;
    *new_deletes=0;
    thrust::device_ptr<int32_t> triangleRemove_ptr(triangleRemove);
    thrust::fill(triangleRemove_ptr,triangleRemove_ptr+numEdges,0);
    hipDeviceSynchronize();
    truss_decompose<<<dimBlock, dimGrid>>>(triangleCount, triangleOffsets, triangleRemove, triangleBuffers1, triangleBuffers2, edge_exists, new_deletes, numEdges, k);
    hipDeviceSynchronize();
    if (*new_deletes==1) {
    update_triangles<<<dimBlock, dimGrid>>>(triangleCount, triangleOffsets, triangleRemove, triangleBuffers1, triangleBuffers2, numEdges);
    hipDeviceSynchronize();
    }
  }
  // std::cout << "Triangle Count" << std::endl;
	int32_t totalCount = 0;
	// for (int32_t i = 0; i < numEdges; ++i) {
	// 	std::cout << triangleCount[i] << ' ';
	// 	totalCount += triangleCount[i];
	// }
  std::cout << "totalCount: " << totalCount << std::endl;

  std::cout << "kmax = " << k << std::endl;
  
  hipFree(edgeSrc_device);
  hipFree(edgeDst_device);
  hipFree(rowPtr_device);
  hipFree(triangleCount);
  hipFree(triangleOffsets);
  return 0;
}
