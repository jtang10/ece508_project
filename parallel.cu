#include "hip/hip_runtime.h"
#include <vector>
#include <list>
#include <cassert>
#include <algorithm>
#include "util.cpp"
#include "edge_list_file.hpp"
#include "coo-impl.hpp"
#include <chrono>
#include <thrust/scan.h>

__global__ static void count_triangle(uint64_t *triangleCount,        //!< per-edge triangle counts
                                      const uint32_t *const edgeSrc,  //!< node ids for edge srcs
                                      const uint32_t *const edgeDst,  //!< node ids for edge dsts
                                      const uint32_t *const rowPtr,   //!< source node offsets in edgeDst
                                      const uint32_t numEdges         //!< how many edges to count triangles for
) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  for(uint32_t i = idx; i < numEdges; i += blockDim.x * gridDim.x) {
    // Determine the source and destination node for the edge
    uint32_t u = edgeSrc[idx];
    uint32_t v = edgeDst[idx];

    // Use the row pointer array to determine the start and end of the neighbor list in the column index array
    uint32_t u_ptr = rowPtr[u];
    uint32_t v_ptr = rowPtr[v];

    uint32_t u_end = rowPtr[u + 1];
    uint32_t v_end = rowPtr[v + 1];

    uint32_t w1 = edgeDst[u_ptr];
    uint32_t w2 = edgeDst[v_ptr];

    // Determine how many elements of those two arrays are common
    while ((u_ptr < u_end) && (v_ptr < v_end)) {
      if (w1 < w2) {
        w1 = edgeDst[++u_ptr];
      } else if (w1 > w2) {
        w2 = edgeDst[++v_ptr];
      } else {
        w1 = edgeDst[++u_ptr];
        w2 = edgeDst[++v_ptr];
        triangleCount[idx]++;
      }
    }

    // if (triangleCount[idx]) {
    //   printf("Thread %d output: %d", threadIdx.x, triangleCount[idx]);
    // }
  }
}

__global__ static void enum_triangle()

int main(int argc, char * argv[]) {
  std::string test_filename;	
  if (argv[1] == NULL) {
      test_filename = "./data/test3.bel";
  } else {
    test_filename = argv[1];
  }
  EdgeListFile test_file(test_filename);

  // get the total number of edges in the file.
  std::vector<EdgeTy<uint32_t>> edges;
  uint32_t size = getNumEdges(test_filename);
  // std::cout << "Numbers of edges in the file : " << size << std::endl;

  // read the bel file into the EdgeListFile
  uint32_t numEdge = test_file.get_edges(edges, size);
  // std::cout << "Confirmed read edges: " << numEdge << std::endl;
  
  COO<uint32_t> coo_test = COO<uint32_t>::from_edges<std::vector<EdgeTy<uint32_t>>::iterator>(edges.begin(), edges.end());
  COOView<uint32_t> test_view = coo_test.view();
  
  uint32_t numEdges = test_view.nnz();
  uint32_t numRows = test_view.num_rows();
  // vector<uint32_t> triangleCount(numEdges);  // keep track of the number of triangles for each edge
  // vector<vector<pair<uint32_t, uint32_t>>> triangleList(numEdges); // keep track of the triangle edges for each edge
  assert(size == numEdge && "number of edges from bel file size does not equal confirmed read edges.");
  assert(numEdge == numEdges && "number of edges in COO does not equal confirmed read edges");
  std::cout << "numEdges from nnz: " << numEdges << std::endl;


	uint32_t* edgeSrc_device = nullptr;
  uint32_t* edgeDst_device = nullptr;
	uint32_t* rowPtr_device = nullptr;
	uint64_t* triangleCount = nullptr;
	uint64_t* triangleCountScan = nullptr;

	//allocate necessary memory
	hipMalloc(&edgeSrc_device, numEdges*sizeof(uint32_t));
	hipMalloc(&edgeDst_device, numEdges*sizeof(uint32_t));
	hipMalloc(&rowPtr_device, (numRows+1)*sizeof(uint32_t));
	hipMallocManaged(&triangleCount, numEdges*sizeof(uint64_t));
	hipMallocManaged(&triangleCountScan, numEdges*sizeof(uint64_t));
	//copy over data
	hipMemcpy(edgeSrc_device, test_view.row_ind(), numEdges*sizeof(uint32_t),hipMemcpyHostToDevice);
	hipMemcpy(edgeDst_device, test_view.col_ind(), numEdges*sizeof(uint32_t),hipMemcpyHostToDevice);
	hipMemcpy(rowPtr_device, test_view.row_ptr(), (numRows+1)*sizeof(uint32_t),hipMemcpyHostToDevice);
  //call triangle_count
  // chrono::steady_clock::time_point begin = chrono::steady_clock::now();
  dim3 dimBlock(512);
  dim3 dimGrid (ceil(numEdges * 1.0 / dimBlock.x));
	count_triangle<<<dimBlock, dimGrid>>>(triangleCount, edgeSrc_device, edgeDst_device, rowPtr_device, numEdges);
	hipDeviceSynchronize();
  // chrono::steady_clock::time_point end= chrono::steady_clock::now();
  // std::cout << "Triangle count time = " << chrono::duration_cast<chrono::microseconds> (end - begin).count() << " us" << std::endl;
	std::cout << "Triangle Count" << std::endl;
	uint64_t totalCount = 0;
	for (uint32_t i = 0; i < numEdges; ++i) {
		std::cout << triangleCount[i] << ' ';
		totalCount += triangleCount[i];
	}
  std::cout << "totalCount: " << totalCount << std::endl;

  thrust::inclusive_scan(triangleCount, triangleCount+numEdges, triangleCountScan);
  std::cout << "Triangle Count Scan" << std::endl;
	for (uint32_t i = 0; i < numEdges; ++i) {
		std::cout << triangleCountScan[i] << ' ';
	}
  
  hipFree(edgeSrc_device);
  hipFree(edgeDst_device);
  hipFree(rowPtr_device);
  hipFree(triangleCount);

  return 0;
}
